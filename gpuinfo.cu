#include "hip/hip_runtime.h"
#include "gpuinfo.h"
#include <hip/hip_runtime.h>


#define BUF_BYTES 256



GPUInfo* CreateGPUInfo()
{
   return new GPUInfo;
}


GPUInfo::GPUInfo()
{
   hipError_t error_id = hipGetDeviceCount(&m_nGPUS);

   if(error_id != hipSuccess)
   {
      m_nGPUS = 0;
   }
}

std::vector<string> GPUInfo::GetGPUProps(uint32_t idx)
{
   char buf[128];

   // TODO: verify index is in bounds
   vector<string> strlist;

   sprintf_s(buf, "GPU index %d:", idx);
   strlist.push_back(buf);

#if 1
   int32_t deviceCount = 0;
   hipError_t error_id = hipGetDeviceCount(&deviceCount);

   if (error_id != hipSuccess)
   {
      sprintf_s(buf, "cuda error obtaining device count: %s", hipGetErrorString(error_id));
      strlist.push_back(buf);
      return strlist;
   }

   if((int32_t)idx > deviceCount - 1 )
   {
      sprintf_s(buf, "Invalid index %d.  cuda device count: %d", idx, deviceCount);
      strlist.push_back(buf);
      return strlist;
   }

   int32_t dev = idx;
   int32_t driverVersion = 0, runtimeVersion = 0;

   hipSetDevice(dev);
   hipDeviceProp_t deviceProp;
   hipGetDeviceProperties(&deviceProp, dev);

   sprintf_s(buf, "\nDevice %d: \"%s\"\n", dev, deviceProp.name);
   strlist.push_back(buf);

   // Console log
   hipDriverGetVersion(&driverVersion);
   hipRuntimeGetVersion(&runtimeVersion);
   sprintf_s(buf, "  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, (runtimeVersion%100)/10);
   strlist.push_back(buf);
   sprintf_s(buf, "  CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);
   strlist.push_back(buf);

   sprintf_s(buf, "  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
          (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
   strlist.push_back(buf);

//   sprintf_s(buf, "  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
//         deviceProp.multiProcessorCount,
//         _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
//         _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);
//   strlist.push_back(buf);
   sprintf_s(buf, "  GPU Max Clock rate:                            %.0f MHz (%0.2f GHz)\n", deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
   strlist.push_back(buf);


#if CUDART_VERSION >= 5000
   // This is supported in CUDA 5.0 (runtime API device properties)
   sprintf_s(buf, "  Memory Clock rate:                             %.0f Mhz\n", deviceProp.memoryClockRate * 1e-3f);
   strlist.push_back(buf);
   sprintf_s(buf, "  Memory Bus Width:                              %d-bit\n",   deviceProp.memoryBusWidth);
   strlist.push_back(buf);

   if (deviceProp.l2CacheSize)
   {
      sprintf_s(buf, "  L2 Cache Size:                                 %d bytes\n", deviceProp.l2CacheSize);
      strlist.push_back(buf);
   }

#else
   // This only available in CUDA 4.0-4.2 (but these were only exposed in the CUDA Driver API)
   int memoryClock;
   getCudaAttribute<int>(&memoryClock, hipDeviceAttributeMemoryClockRate, dev);
   sprintf_s(buf, "  Memory Clock rate:                             %.0f Mhz\n", memoryClock * 1e-3f);
   strlist.push_back(buf);
   int memBusWidth;
   getCudaAttribute<int>(&memBusWidth, hipDeviceAttributeMemoryBusWidth, dev);
   sprintf_s(buf, "  Memory Bus Width:                              %d-bit\n", memBusWidth);
   strlist.push_back(buf);
   int L2CacheSize;
   getCudaAttribute<int>(&L2CacheSize, hipDeviceAttributeL2CacheSize, dev);

   if (L2CacheSize)
   {
      sprintf_s(buf, "  L2 Cache Size:                                 %d bytes\n", L2CacheSize);
      strlist.push_back(buf);
   }

#endif

   sprintf_s(buf, "  Maximum Texture Dimension Size (x,y,z)         1D=(%d), 2D=(%d, %d), 3D=(%d, %d, %d)\n",
         deviceProp.maxTexture1D   , deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1],
         deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
   strlist.push_back(buf);
   sprintf_s(buf, "  Maximum Layered 1D Texture Size, (num) layers  1D=(%d), %d layers\n",
         deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1]);
   strlist.push_back(buf);
   sprintf_s(buf, "  Maximum Layered 2D Texture Size, (num) layers  2D=(%d, %d), %d layers\n",
         deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1], deviceProp.maxTexture2DLayered[2]);
   strlist.push_back(buf);


   sprintf_s(buf, "  Total amount of constant memory:               %lu bytes\n", deviceProp.totalConstMem);
   strlist.push_back(buf);
   sprintf_s(buf, "  Total amount of shared memory per block:       %lu bytes\n", deviceProp.sharedMemPerBlock);
   strlist.push_back(buf);
   sprintf_s(buf, "  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
   strlist.push_back(buf);
   sprintf_s(buf, "  Warp size:                                     %d\n", deviceProp.warpSize);
   strlist.push_back(buf);
   sprintf_s(buf, "  Maximum number of threads per multiprocessor:  %d\n", deviceProp.maxThreadsPerMultiProcessor);
   strlist.push_back(buf);
   sprintf_s(buf, "  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
   strlist.push_back(buf);
   sprintf_s(buf, "  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
         deviceProp.maxThreadsDim[0],
         deviceProp.maxThreadsDim[1],
         deviceProp.maxThreadsDim[2]);
   strlist.push_back(buf);
   sprintf_s(buf, "  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
         deviceProp.maxGridSize[0],
         deviceProp.maxGridSize[1],
         deviceProp.maxGridSize[2]);
   strlist.push_back(buf);
   sprintf_s(buf, "  Maximum memory pitch:                          %lu bytes\n", deviceProp.memPitch);
   strlist.push_back(buf);
   sprintf_s(buf, "  Texture alignment:                             %lu bytes\n", deviceProp.textureAlignment);
   strlist.push_back(buf);
   sprintf_s(buf, "  Concurrent copy and kernel execution:          %s with %d copy engine(s)\n", (deviceProp.deviceOverlap ? "Yes" : "No"), deviceProp.asyncEngineCount);
   strlist.push_back(buf);
   sprintf_s(buf, "  Run time limit on kernels:                     %s\n", deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
   strlist.push_back(buf);
   sprintf_s(buf, "  Integrated GPU sharing Host Memory:            %s\n", deviceProp.integrated ? "Yes" : "No");
   strlist.push_back(buf);
   sprintf_s(buf, "  Support host page-locked memory mapping:       %s\n", deviceProp.canMapHostMemory ? "Yes" : "No");
   strlist.push_back(buf);
   sprintf_s(buf, "  Alignment requirement for Surfaces:            %s\n", deviceProp.surfaceAlignment ? "Yes" : "No");
   strlist.push_back(buf);
   sprintf_s(buf, "  Device has ECC support:                        %s\n", deviceProp.ECCEnabled ? "Enabled" : "Disabled");
   strlist.push_back(buf);
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
   sprintf_s(buf, "  CUDA Device Driver Mode (TCC or WDDM):         %s\n", deviceProp.tccDriver ? "TCC (Tesla Compute Cluster Driver)" : "WDDM (Windows Display Driver Model)");
   strlist.push_back(buf);
#endif
   sprintf_s(buf, "  Device supports Unified Addressing (UVA):      %s\n", deviceProp.unifiedAddressing ? "Yes" : "No");
   strlist.push_back(buf);
   sprintf_s(buf, "  Device PCI Domain ID / Bus ID / location ID:   %d / %d / %d\n", deviceProp.pciDomainID, deviceProp.pciBusID, deviceProp.pciDeviceID);
   strlist.push_back(buf);

   const char *sComputeMode[] =
   {
      "Default (multiple host threads can use ::hipSetDevice() with device simultaneously)",
      "Exclusive (only one host thread in one process is able to use ::hipSetDevice() with this device)",
      "Prohibited (no host thread can use ::hipSetDevice() with this device)",
      "Exclusive Process (many threads in one process is able to use ::hipSetDevice() with this device)",
      "Unknown",
      NULL
   };
   sprintf_s(buf, "  Compute Mode:\n");
   strlist.push_back(buf);
   if( 6 > deviceProp.computeMode)
   {
      sprintf_s(buf, "     < %s >\n", sComputeMode[deviceProp.computeMode]);
      strlist.push_back(buf);
   }
   else
   {
      sprintf_s(buf, "     Unexpected computeMode %d\n",deviceProp.computeMode);
      strlist.push_back(buf);
   }

    // If there are 2 or more GP
   //hipDeviceProp_t prop[64];
   //checkCudaErrors(hipGetDeviceProperties(&prop[dev], dev));

      // Only boards based on Fermi or later can support P2P
   if ((deviceProp.major >= 2)
 #if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
          // on Windows (64-bit), the Tesla Compute Cluster driver for windows must be enabled to support this
          && deviceProp.tccDriver
#endif
         )
      {
          // This is an array of P2P capable GPUs
         strlist.push_back("Peer to peer access capable!");
      }


    // csv masterlog info
    // *****************************
    // exe and CUDA driver name
    sprintf_s(buf, "\n");
    std::string sProfileString = "deviceQuery, CUDA Driver = CUDART";
    char cTemp[16];

    // driver version
    sProfileString += ", CUDA Driver Version = ";
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    sprintf_s(cTemp, "%d.%d", driverVersion/1000, (driverVersion%100)/10);
#else
    sprintf_s(cTemp, "%d.%d", driverVersion/1000, (driverVersion%100)/10);
#endif
    sProfileString +=  cTemp;

    // Runtime version
    sProfileString += ", CUDA Runtime Version = ";
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    sprintf_s(cTemp, "%d.%d", runtimeVersion/1000, (runtimeVersion%100)/10);
#else
    ssprintf_s(buf, cTemp, "%d.%d", runtimeVersion/1000, (runtimeVersion%100)/10);
#endif
    sProfileString +=  cTemp;

    // Device count
    sProfileString += ", NumDevs = ";
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    sprintf_s(cTemp, "%d", deviceCount);
#else
    ssprintf_s(buf, cTemp, "%d", deviceCount);
#endif
    sProfileString += cTemp;


    sProfileString += "\n";
    strlist.push_back(sProfileString);
#endif

    return strlist;
}

void* GPUInfo::AllocMem(uint32_t nMB)
{
   hipError_t ret = hipMalloc(&m_dvcBufPtr, nMB * 0x00100000);

   if(hipSuccess != ret)
   {
      m_dvcBufPtr = nullptr;
   }

   return m_dvcBufPtr;

}
